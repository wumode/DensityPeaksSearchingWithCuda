#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""

#include "mex.h"
#include "gpu\mxGPUArray.h"

//__constant__ int index_offset3x3[9];


__global__ void computeDeltaKernel(int* rhoMat, int* pxMat, int* pyMat, float* delta, int* signMatU, int* index_offset3x3, int height, int width, int d_height)
{
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int x = idx / height;
    int y = idx % height;
    if (0<x&&x<width-1 &&0<y&&y<height-1) {
        //int rho = rhoMat[idx];
        int rhoVector[] = { rhoMat[idx - height - 1], rhoMat[idx - height], rhoMat[idx - height + 1],
                            rhoMat[idx - 1],          rhoMat[idx],          rhoMat[idx + 1],
                            rhoMat[idx + height - 1], rhoMat[idx + height], rhoMat[idx + height + 1] };
        int rho_max = 0;
        unsigned char max_index = 0;
        for (int i = 0; i < 4; ++i) {
            if (rhoVector[i] >= rho_max) {
                rho_max = rhoVector[i];
                max_index = i;
            }
        }
        for (int i = 4; i < 9; ++i) {
            if (rhoVector[i] > rho_max) {
                rho_max = rhoVector[i];
                max_index = i;
            }
        }
        if (max_index == 4) {
            signMatU[idx] = 1;
        }
        else {
            signMatU[idx] = 0;
            int idmax = idx + index_offset3x3[max_index];
            float dist = (float)((pxMat[idmax] - pxMat[idx]) * (pxMat[idmax] - pxMat[idx]) + 
                (pyMat[idmax] - pyMat[idx]) * (pyMat[idmax] - pyMat[idx]));
            dist = sqrt(dist);
            int d_idx = d_height * (pxMat[idx]-1) + pyMat[idx]-1;
            delta[d_idx] = dist;
        }
    }
}


hipError_t parallelSelectionWithCuda(int* rhoMat, int* pxMat, int* pyMat, float* delta, int* signMatU, int height, int width, int d_height, int d_width) {
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    int* dev_rhoMat = 0;
    int* dev_pxMat = 0;
    int* dev_pyMat = 0;
    float* dev_delta = 0;
    int* dev_signMatU = 0;
    int* dev_index_offset3x3 = 0;
    unsigned int size =height*width ;
    unsigned int dsize = d_height*d_width ;
    int index_offset3x3[] = { -height - 1, -height, -height + 1, -1, 0, 1, height - 1, height, height + 1 };
    //hipMemcpyToSymbol(HIP_SYMBOL(index_offset3x3), index_offset3x3_tmp, sizeof(int) * 9);

    cudaStatus = hipMalloc((void**)&dev_index_offset3x3, 9 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMalloc dev_index_offset3x3 %d bytes failed!\n", sizeof(int) * 9 );
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rhoMat, size* sizeof(int));
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMalloc dev_rhoMat %d bytes failed!\n", size * sizeof(int));
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pxMat, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMalloc dev_pxMat failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pyMat, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMalloc dev_pyMat failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_delta, dsize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMalloc dev_delta failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_signMatU, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMalloc dev_signMatU failed!");
        goto Error;
    }

    dim3 blockSize(256);
    dim3 gridSize((int)((size + blockSize.x - 1) / blockSize.x));

    //mexPrintf("gridSize: %d\nblockSize: %d\n", gridSize.x, blockSize.x);
    // Copy input vectors from host memory to GPU buffers.

    cudaStatus = hipMemcpy(dev_index_offset3x3, index_offset3x3, 9 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMemcpy dev_index_offset3x3 failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rhoMat, rhoMat, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMemcpy dev_rhoMat failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_pxMat, pxMat, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_pyMat, pyMat, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_delta, delta, dsize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        mexPrintf("hipMemcpy failed!");
        goto Error;
    }
    computeDeltaKernel << <gridSize, blockSize >> > (dev_rhoMat, dev_pxMat, dev_pyMat, dev_delta, dev_signMatU, dev_index_offset3x3, height, width, d_height);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        mexPrintf( "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipDeviceSynchronize returned error code %d after launching computeDeltaKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(signMatU, dev_signMatU, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(delta, dev_delta, dsize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        mexPrintf( "hipMemcpy failed!");
        goto Error;
    }
    return cudaStatus;
Error:
    hipFree(dev_rhoMat);
    hipFree(dev_pxMat);
    hipFree(dev_pyMat);
    hipFree(dev_delta);
    hipFree(dev_signMatU);
    hipFree(dev_index_offset3x3);
    return cudaStatus;
}


void mexFunction(int nlhs, mxArray* plhs[], int nrhs, mxArray const *prhs[])
{
    if (nrhs != 5) {
        mexErrMsgTxt("Wrong number of input arguments.\n");
    }
    if (nlhs > 5) {
        mexErrMsgTxt("Too many output argumnents.\n");
    }

    int* rhoMat = (int*)mxGetPr(prhs[0]);
    int* pxMat = (int*)mxGetPr(prhs[1]);
    int* pyMat = (int*)mxGetPr(prhs[2]);
    float* delta = (float*)mxGetPr(prhs[3]);

    int M = mxGetM(prhs[0]);
    int N = mxGetN(prhs[0]);
    int* sign = (int*)malloc(sizeof(int) * M * N);

    int dM = mxGetM(prhs[3]);
    int dN = mxGetN(prhs[3]);
    //mexPrintf("%dx%d\n", dM, dN);
    //mexPrintf("%dx%d\n", M, N);
    hipError_t cudaStatus = parallelSelectionWithCuda(rhoMat, pxMat, pyMat, delta, sign, M, N, dM, dN);
    if (cudaStatus != hipSuccess) {
        mexErrMsgTxt("Exit with CUDA error.\n");
    }

    plhs[0] = mxCreateDoubleMatrix(M, N, mxREAL);
    plhs[1] = mxCreateDoubleMatrix(dM, dN, mxREAL);

    double* sign_out = mxGetPr(plhs[0]);
    double* delta_out = mxGetPr(plhs[1]);


    for (int i = 0; i < dM*dN; ++i) {
        delta_out[i] = delta[i];
    }
    for (int i = 0; i < M*N; ++i) {
        sign_out[i] = sign[i];
    }
    free(sign);
}